#include "hip/hip_runtime.h"
// This file is part of Eigen, a lightweight C++ template library
// for linear algebra.
//
// Copyright (C) 2014 Benoit Steiner <benoit.steiner.goog@gmail.com>
//
// This Source Code Form is subject to the terms of the Mozilla
// Public License v. 2.0. If a copy of the MPL was not distributed
// with this file, You can obtain one at http://mozilla.org/MPL/2.0/.

#define EIGEN_TEST_NO_LONGDOUBLE
#define EIGEN_TEST_NO_COMPLEX
#define EIGEN_TEST_FUNC cxx11_tensor_cuda
#define EIGEN_DEFAULT_DENSE_INDEX_TYPE int
#define EIGEN_USE_GPU


#include "main.h"
#include <unsupported/Eigen/CXX11/Tensor>

using Eigen::Tensor;

void test_cuda_elementwise_small() {
  Tensor<float, 1> in1(Eigen::array<int, 1>(2));
  Tensor<float, 1> in2(Eigen::array<int, 1>(2));
  Tensor<float, 1> out(Eigen::array<int, 1>(2));
  in1.setRandom();
  in2.setRandom();

  std::size_t in1_bytes = in1.size() * sizeof(float);
  std::size_t in2_bytes = in2.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_in1;
  float* d_in2;
  float* d_out;
  hipMalloc((void**)(&d_in1), in1_bytes);
  hipMalloc((void**)(&d_in2), in2_bytes);
  hipMalloc((void**)(&d_out), out_bytes);

  hipMemcpy(d_in1, in1.data(), in1_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_in2, in2.data(), in2_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_in1(
      d_in1, Eigen::array<int, 1>(2));
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_in2(
      d_in2, Eigen::array<int, 1>(2));
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_out(
      d_out, Eigen::array<int, 1>(2));

  gpu_out.device(gpu_device) = gpu_in1 + gpu_in2;

  assert(hipMemcpyAsync(out.data(), d_out, out_bytes, hipMemcpyDeviceToHost,
                         gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 2; ++i) {
    VERIFY_IS_APPROX(
        out(Eigen::array<int, 1>(i)),
        in1(Eigen::array<int, 1>(i)) + in2(Eigen::array<int, 1>(i)));
  }

  hipFree(d_in1);
  hipFree(d_in2);
  hipFree(d_out);
}

void test_cuda_elementwise()
{
  Tensor<float, 3> in1(Eigen::array<int, 3>(72,53,97));
  Tensor<float, 3> in2(Eigen::array<int, 3>(72,53,97));
  Tensor<float, 3> in3(Eigen::array<int, 3>(72,53,97));
  Tensor<float, 3> out(Eigen::array<int, 3>(72,53,97));
  in1.setRandom();
  in2.setRandom();
  in3.setRandom();

  std::size_t in1_bytes = in1.size() * sizeof(float);
  std::size_t in2_bytes = in2.size() * sizeof(float);
  std::size_t in3_bytes = in3.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_in1;
  float* d_in2;
  float* d_in3;
  float* d_out;
  hipMalloc((void**)(&d_in1), in1_bytes);
  hipMalloc((void**)(&d_in2), in2_bytes);
  hipMalloc((void**)(&d_in3), in3_bytes);
  hipMalloc((void**)(&d_out), out_bytes);

  hipMemcpy(d_in1, in1.data(), in1_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_in2, in2.data(), in2_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_in3, in3.data(), in3_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_in1(d_in1, Eigen::array<int, 3>(72,53,97));
  Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_in2(d_in2, Eigen::array<int, 3>(72,53,97));
  Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_in3(d_in3, Eigen::array<int, 3>(72,53,97));
  Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_out(d_out, Eigen::array<int, 3>(72,53,97));

  gpu_out.device(gpu_device) = gpu_in1 + gpu_in2 * gpu_in3;

  assert(hipMemcpyAsync(out.data(), d_out, out_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 72; ++i) {
    for (int j = 0; j < 53; ++j) {
      for (int k = 0; k < 97; ++k) {
        VERIFY_IS_APPROX(out(Eigen::array<int, 3>(i,j,k)), in1(Eigen::array<int, 3>(i,j,k)) + in2(Eigen::array<int, 3>(i,j,k)) * in3(Eigen::array<int, 3>(i,j,k)));
      }
    }
  }

  hipFree(d_in1);
  hipFree(d_in2);
  hipFree(d_in3);
  hipFree(d_out);
}

void test_cuda_reduction()
{
  Tensor<float, 4> in1(72,53,97,113);
  Tensor<float, 2> out(72,97);
  in1.setRandom();

  std::size_t in1_bytes = in1.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_in1;
  float* d_out;
  hipMalloc((void**)(&d_in1), in1_bytes);
  hipMalloc((void**)(&d_out), out_bytes);

  hipMemcpy(d_in1, in1.data(), in1_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 4> > gpu_in1(d_in1, 72,53,97,113);
  Eigen::TensorMap<Eigen::Tensor<float, 2> > gpu_out(d_out, 72,97);

  array<int, 2> reduction_axis;
  reduction_axis[0] = 1;
  reduction_axis[1] = 3;

  gpu_out.device(gpu_device) = gpu_in1.maximum(reduction_axis);

  assert(hipMemcpyAsync(out.data(), d_out, out_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 72; ++i) {
    for (int j = 0; j < 97; ++j) {
      float expected = 0;
      for (int k = 0; k < 53; ++k) {
        for (int l = 0; l < 113; ++l) {
          expected =
              std::max<float>(expected, in1(i, k, j, l));
        }
      }
      VERIFY_IS_APPROX(out(i,j), expected);
    }
  }

  hipFree(d_in1);
  hipFree(d_out);
}

template<int DataLayout>
void test_cuda_contraction()
{
  // with these dimensions, the output has 300 * 140 elements, which is
  // more than 30 * 1024, which is the number of threads in blocks on
  // a 15 SM GK110 GPU
  Tensor<float, 4, DataLayout> t_left(6, 50, 3, 31);
  Tensor<float, 5, DataLayout> t_right(Eigen::array<int, 5>(3, 31, 7, 20, 1));
  Tensor<float, 5, DataLayout> t_result(Eigen::array<int, 5>(6, 50, 7, 20, 1));

  t_left.setRandom();
  t_right.setRandom();

  std::size_t t_left_bytes = t_left.size()  * sizeof(float);
  std::size_t t_right_bytes = t_right.size() * sizeof(float);
  std::size_t t_result_bytes = t_result.size() * sizeof(float);

  float* d_t_left;
  float* d_t_right;
  float* d_t_result;

  hipMalloc((void**)(&d_t_left), t_left_bytes);
  hipMalloc((void**)(&d_t_right), t_right_bytes);
  hipMalloc((void**)(&d_t_result), t_result_bytes);

  hipMemcpy(d_t_left, t_left.data(), t_left_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_t_right, t_right.data(), t_right_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 4, DataLayout> > gpu_t_left(d_t_left, 6, 50, 3, 31);
  Eigen::TensorMap<Eigen::Tensor<float, 5, DataLayout> > gpu_t_right(d_t_right, 3, 31, 7, 20, 1);
  Eigen::TensorMap<Eigen::Tensor<float, 5, DataLayout> > gpu_t_result(d_t_result, 6, 50, 7, 20, 1);

  typedef Eigen::Map<Eigen::Matrix<float, Dynamic, Dynamic, DataLayout> > MapXf;
  MapXf m_left(t_left.data(), 300, 93);
  MapXf m_right(t_right.data(), 93, 140);
  Eigen::Matrix<float, Dynamic, Dynamic, DataLayout> m_result(300, 140);

  typedef Tensor<float, 1>::DimensionPair DimPair;
  Eigen::array<DimPair, 2> dims;
  dims[0] = DimPair(2, 0);
  dims[1] = DimPair(3, 1);

  m_result = m_left * m_right;
  gpu_t_result.device(gpu_device) = gpu_t_left.contract(gpu_t_right, dims);

  hipMemcpy(t_result.data(), d_t_result, t_result_bytes, hipMemcpyDeviceToHost);

  for (size_t i = 0; i < t_result.dimensions().TotalSize(); i++) {
    if (fabs(t_result.data()[i] - m_result.data()[i]) >= 1e-4) {
      std::cout << "mismatch detected at index " << i << ": " << t_result.data()[i] << " vs " <<  m_result.data()[i] << std::endl;
      assert(false);
    }
  }

  hipFree(d_t_left);
  hipFree(d_t_right);
  hipFree(d_t_result);
}

template<int DataLayout>
void test_cuda_convolution_1d()
{
  Tensor<float, 4, DataLayout> input(74,37,11,137);
  Tensor<float, 1, DataLayout> kernel(4);
  Tensor<float, 4, DataLayout> out(74,34,11,137);
  input = input.constant(10.0f) + input.random();
  kernel = kernel.constant(7.0f) + kernel.random();

  std::size_t input_bytes = input.size() * sizeof(float);
  std::size_t kernel_bytes = kernel.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_input;
  float* d_kernel;
  float* d_out;
  hipMalloc((void**)(&d_input), input_bytes);
  hipMalloc((void**)(&d_kernel), kernel_bytes);
  hipMalloc((void**)(&d_out), out_bytes);

  hipMemcpy(d_input, input.data(), input_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_kernel, kernel.data(), kernel_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 4, DataLayout> > gpu_input(d_input, 74,37,11,137);
  Eigen::TensorMap<Eigen::Tensor<float, 1, DataLayout> > gpu_kernel(d_kernel, 4);
  Eigen::TensorMap<Eigen::Tensor<float, 4, DataLayout> > gpu_out(d_out, 74,34,11,137);

  Eigen::array<int, 1> dims(1);
  gpu_out.device(gpu_device) = gpu_input.convolve(gpu_kernel, dims);

  assert(hipMemcpyAsync(out.data(), d_out, out_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 74; ++i) {
    for (int j = 0; j < 34; ++j) {
      for (int k = 0; k < 11; ++k) {
        for (int l = 0; l < 137; ++l) {
          const float result = out(i,j,k,l);
          const float expected = input(i,j+0,k,l) * kernel(0) + input(i,j+1,k,l) * kernel(1) +
                                 input(i,j+2,k,l) * kernel(2) + input(i,j+3,k,l) * kernel(3);
          VERIFY_IS_APPROX(result, expected);
        }
      }
    }
  }

  hipFree(d_input);
  hipFree(d_kernel);
  hipFree(d_out);
}

void test_cuda_convolution_inner_dim_col_major_1d()
{
  Tensor<float, 4, ColMajor> input(74,9,11,7);
  Tensor<float, 1, ColMajor> kernel(4);
  Tensor<float, 4, ColMajor> out(71,9,11,7);
  input = input.constant(10.0f) + input.random();
  kernel = kernel.constant(7.0f) + kernel.random();

  std::size_t input_bytes = input.size() * sizeof(float);
  std::size_t kernel_bytes = kernel.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_input;
  float* d_kernel;
  float* d_out;
  hipMalloc((void**)(&d_input), input_bytes);
  hipMalloc((void**)(&d_kernel), kernel_bytes);
  hipMalloc((void**)(&d_out), out_bytes);

  hipMemcpy(d_input, input.data(), input_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_kernel, kernel.data(), kernel_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 4, ColMajor> > gpu_input(d_input,74,9,11,7);
  Eigen::TensorMap<Eigen::Tensor<float, 1, ColMajor> > gpu_kernel(d_kernel,4);
  Eigen::TensorMap<Eigen::Tensor<float, 4, ColMajor> > gpu_out(d_out,71,9,11,7);

  Eigen::array<int, 1> dims(0);
  gpu_out.device(gpu_device) = gpu_input.convolve(gpu_kernel, dims);

  assert(hipMemcpyAsync(out.data(), d_out, out_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 71; ++i) {
    for (int j = 0; j < 9; ++j) {
      for (int k = 0; k < 11; ++k) {
        for (int l = 0; l < 7; ++l) {
          const float result = out(i,j,k,l);
          const float expected = input(i+0,j,k,l) * kernel(0) + input(i+1,j,k,l) * kernel(1) +
                                 input(i+2,j,k,l) * kernel(2) + input(i+3,j,k,l) * kernel(3);
          VERIFY_IS_APPROX(result, expected);
        }
      }
    }
  }

  hipFree(d_input);
  hipFree(d_kernel);
  hipFree(d_out);
}

void test_cuda_convolution_inner_dim_row_major_1d()
{
  Tensor<float, 4, RowMajor> input(7,9,11,74);
  Tensor<float, 1, RowMajor> kernel(4);
  Tensor<float, 4, RowMajor> out(7,9,11,71);
  input = input.constant(10.0f) + input.random();
  kernel = kernel.constant(7.0f) + kernel.random();

  std::size_t input_bytes = input.size() * sizeof(float);
  std::size_t kernel_bytes = kernel.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_input;
  float* d_kernel;
  float* d_out;
  hipMalloc((void**)(&d_input), input_bytes);
  hipMalloc((void**)(&d_kernel), kernel_bytes);
  hipMalloc((void**)(&d_out), out_bytes);

  hipMemcpy(d_input, input.data(), input_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_kernel, kernel.data(), kernel_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 4, RowMajor> > gpu_input(d_input, 7,9,11,74);
  Eigen::TensorMap<Eigen::Tensor<float, 1, RowMajor> > gpu_kernel(d_kernel, 4);
  Eigen::TensorMap<Eigen::Tensor<float, 4, RowMajor> > gpu_out(d_out, 7,9,11,71);

  Eigen::array<int, 1> dims(3);
  gpu_out.device(gpu_device) = gpu_input.convolve(gpu_kernel, dims);

  assert(hipMemcpyAsync(out.data(), d_out, out_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 7; ++i) {
    for (int j = 0; j < 9; ++j) {
      for (int k = 0; k < 11; ++k) {
        for (int l = 0; l < 71; ++l) {
          const float result = out(i,j,k,l);
          const float expected = input(i,j,k,l+0) * kernel(0) + input(i,j,k,l+1) * kernel(1) +
                                 input(i,j,k,l+2) * kernel(2) + input(i,j,k,l+3) * kernel(3);
          VERIFY_IS_APPROX(result, expected);
        }
      }
    }
  }

  hipFree(d_input);
  hipFree(d_kernel);
  hipFree(d_out);
}

template<int DataLayout>
void test_cuda_convolution_2d()
{
  Tensor<float, 4, DataLayout> input(74,37,11,137);
  Tensor<float, 2, DataLayout> kernel(3,4);
  Tensor<float, 4, DataLayout> out(74,35,8,137);
  input = input.constant(10.0f) + input.random();
  kernel = kernel.constant(7.0f) + kernel.random();

  std::size_t input_bytes = input.size() * sizeof(float);
  std::size_t kernel_bytes = kernel.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_input;
  float* d_kernel;
  float* d_out;
  hipMalloc((void**)(&d_input), input_bytes);
  hipMalloc((void**)(&d_kernel), kernel_bytes);
  hipMalloc((void**)(&d_out), out_bytes);

  hipMemcpy(d_input, input.data(), input_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_kernel, kernel.data(), kernel_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 4, DataLayout> > gpu_input(d_input,74,37,11,137);
  Eigen::TensorMap<Eigen::Tensor<float, 2, DataLayout> > gpu_kernel(d_kernel,3,4);
  Eigen::TensorMap<Eigen::Tensor<float, 4, DataLayout> > gpu_out(d_out,74,35,8,137);

  Eigen::array<int, 2> dims(1,2);
  gpu_out.device(gpu_device) = gpu_input.convolve(gpu_kernel, dims);

  assert(hipMemcpyAsync(out.data(), d_out, out_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 74; ++i) {
    for (int j = 0; j < 35; ++j) {
      for (int k = 0; k < 8; ++k) {
        for (int l = 0; l < 137; ++l) {
          const float result = out(i,j,k,l);
          const float expected = input(i,j+0,k+0,l) * kernel(0,0) +
                                 input(i,j+1,k+0,l) * kernel(1,0) +
                                 input(i,j+2,k+0,l) * kernel(2,0) +
                                 input(i,j+0,k+1,l) * kernel(0,1) +
                                 input(i,j+1,k+1,l) * kernel(1,1) +
                                 input(i,j+2,k+1,l) * kernel(2,1) +
                                 input(i,j+0,k+2,l) * kernel(0,2) +
                                 input(i,j+1,k+2,l) * kernel(1,2) +
                                 input(i,j+2,k+2,l) * kernel(2,2) +
                                 input(i,j+0,k+3,l) * kernel(0,3) +
                                 input(i,j+1,k+3,l) * kernel(1,3) +
                                 input(i,j+2,k+3,l) * kernel(2,3);
          VERIFY_IS_APPROX(result, expected);
        }
      }
    }
  }

  hipFree(d_input);
  hipFree(d_kernel);
  hipFree(d_out);
}

template<int DataLayout>
void test_cuda_convolution_3d()
{
  Tensor<float, 5, DataLayout> input(Eigen::array<int, 5>(74,37,11,137,17));
  Tensor<float, 3, DataLayout> kernel(3,4,2);
  Tensor<float, 5, DataLayout> out(Eigen::array<int, 5>(74,35,8,136,17));
  input = input.constant(10.0f) + input.random();
  kernel = kernel.constant(7.0f) + kernel.random();

  std::size_t input_bytes = input.size() * sizeof(float);
  std::size_t kernel_bytes = kernel.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_input;
  float* d_kernel;
  float* d_out;
  hipMalloc((void**)(&d_input), input_bytes);
  hipMalloc((void**)(&d_kernel), kernel_bytes);
  hipMalloc((void**)(&d_out), out_bytes);

  hipMemcpy(d_input, input.data(), input_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_kernel, kernel.data(), kernel_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;    
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 5, DataLayout> > gpu_input(d_input,74,37,11,137,17);
  Eigen::TensorMap<Eigen::Tensor<float, 3, DataLayout> > gpu_kernel(d_kernel,3,4,2);
  Eigen::TensorMap<Eigen::Tensor<float, 5, DataLayout> > gpu_out(d_out,74,35,8,136,17);

  Eigen::array<int, 3> dims(1,2,3);
  gpu_out.device(gpu_device) = gpu_input.convolve(gpu_kernel, dims);

  assert(hipMemcpyAsync(out.data(), d_out, out_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 74; ++i) {
    for (int j = 0; j < 35; ++j) {
      for (int k = 0; k < 8; ++k) {
        for (int l = 0; l < 136; ++l) {
          for (int m = 0; m < 17; ++m) {
            const float result = out(i,j,k,l,m);
            const float expected = input(i,j+0,k+0,l+0,m) * kernel(0,0,0) +
                                   input(i,j+1,k+0,l+0,m) * kernel(1,0,0) +
                                   input(i,j+2,k+0,l+0,m) * kernel(2,0,0) +
                                   input(i,j+0,k+1,l+0,m) * kernel(0,1,0) +
                                   input(i,j+1,k+1,l+0,m) * kernel(1,1,0) +
                                   input(i,j+2,k+1,l+0,m) * kernel(2,1,0) +
                                   input(i,j+0,k+2,l+0,m) * kernel(0,2,0) +
                                   input(i,j+1,k+2,l+0,m) * kernel(1,2,0) +
                                   input(i,j+2,k+2,l+0,m) * kernel(2,2,0) +
                                   input(i,j+0,k+3,l+0,m) * kernel(0,3,0) +
                                   input(i,j+1,k+3,l+0,m) * kernel(1,3,0) +
                                   input(i,j+2,k+3,l+0,m) * kernel(2,3,0) +
                                   input(i,j+0,k+0,l+1,m) * kernel(0,0,1) +
                                   input(i,j+1,k+0,l+1,m) * kernel(1,0,1) +
                                   input(i,j+2,k+0,l+1,m) * kernel(2,0,1) +
                                   input(i,j+0,k+1,l+1,m) * kernel(0,1,1) +
                                   input(i,j+1,k+1,l+1,m) * kernel(1,1,1) +
                                   input(i,j+2,k+1,l+1,m) * kernel(2,1,1) +
                                   input(i,j+0,k+2,l+1,m) * kernel(0,2,1) +
                                   input(i,j+1,k+2,l+1,m) * kernel(1,2,1) +
                                   input(i,j+2,k+2,l+1,m) * kernel(2,2,1) +
                                   input(i,j+0,k+3,l+1,m) * kernel(0,3,1) +
                                   input(i,j+1,k+3,l+1,m) * kernel(1,3,1) +
                                   input(i,j+2,k+3,l+1,m) * kernel(2,3,1);
            VERIFY_IS_APPROX(result, expected);
          }
        }
      }
    }
  }

  hipFree(d_input);
  hipFree(d_kernel);
  hipFree(d_out);
}


template <typename Scalar>
void test_cuda_lgamma(const Scalar stddev)
{
  Tensor<Scalar, 2> in(72,97);
  in.setRandom();
  in *= in.constant(stddev);
  Tensor<Scalar, 2> out(72,97);
  out.setZero();

  std::size_t bytes = in.size() * sizeof(Scalar);

  Scalar* d_in;
  Scalar* d_out;
  hipMalloc((void**)(&d_in), bytes);
  hipMalloc((void**)(&d_out), bytes);

  hipMemcpy(d_in, in.data(), bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_in(d_in, 72, 97);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_out(d_out, 72, 97);

  gpu_out.device(gpu_device) = gpu_in.lgamma();

  assert(hipMemcpyAsync(out.data(), d_out, bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 72; ++i) {
    for (int j = 0; j < 97; ++j) {
      VERIFY_IS_APPROX(out(i,j), (std::lgamma)(in(i,j)));
    }
  }

  hipFree(d_in);
  hipFree(d_out);
}

template <typename Scalar>
void test_cuda_digamma()
{
  Tensor<Scalar, 1> in(7);
  Tensor<Scalar, 1> out(7);
  Tensor<Scalar, 1> expected_out(7);
  out.setZero();

  in(0) = Scalar(1);
  in(1) = Scalar(1.5);
  in(2) = Scalar(4);
  in(3) = Scalar(-10.5);
  in(4) = Scalar(10000.5);
  in(5) = Scalar(0);
  in(6) = Scalar(-1);

  expected_out(0) = Scalar(-0.5772156649015329);
  expected_out(1) = Scalar(0.03648997397857645);
  expected_out(2) = Scalar(1.2561176684318);
  expected_out(3) = Scalar(2.398239129535781);
  expected_out(4) = Scalar(9.210340372392849);
  expected_out(5) = std::numeric_limits<Scalar>::infinity();
  expected_out(6) = std::numeric_limits<Scalar>::infinity();

  std::size_t bytes = in.size() * sizeof(Scalar);

  Scalar* d_in;
  Scalar* d_out;
  hipMalloc((void**)(&d_in), bytes);
  hipMalloc((void**)(&d_out), bytes);

  hipMemcpy(d_in, in.data(), bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in(d_in, 7);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_out(d_out, 7);

  gpu_out.device(gpu_device) = gpu_in.digamma();

  assert(hipMemcpyAsync(out.data(), d_out, bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 5; ++i) {
    VERIFY_IS_APPROX(out(i), expected_out(i));
  }
  for (int i = 5; i < 7; ++i) {
    VERIFY_IS_EQUAL(out(i), expected_out(i));
  }
}

template <typename Scalar>
void test_cuda_igamma()
{
  Tensor<Scalar, 2> a(6, 6);
  Tensor<Scalar, 2> x(6, 6);
  Tensor<Scalar, 2> out(6, 6);
  out.setZero();

  Scalar a_s[] = {Scalar(0), Scalar(1), Scalar(1.5), Scalar(4), Scalar(0.0001), Scalar(1000.5)};
  Scalar x_s[] = {Scalar(0), Scalar(1), Scalar(1.5), Scalar(4), Scalar(0.0001), Scalar(1000.5)};

  for (int i = 0; i < 6; ++i) {
    for (int j = 0; j < 6; ++j) {
      a(i, j) = a_s[i];
      x(i, j) = x_s[j];
    }
  }

  Scalar nan = std::numeric_limits<Scalar>::quiet_NaN();
  Scalar igamma_s[][6] = {{0.0, nan, nan, nan, nan, nan},
                          {0.0, 0.6321205588285578, 0.7768698398515702,
                           0.9816843611112658, 9.999500016666262e-05, 1.0},
                          {0.0, 0.4275932955291202, 0.608374823728911,
                           0.9539882943107686, 7.522076445089201e-07, 1.0},
                          {0.0, 0.01898815687615381, 0.06564245437845008,
                           0.5665298796332909, 4.166333347221828e-18, 1.0},
                          {0.0, 0.9999780593618628, 0.9999899967080838,
                           0.9999996219837988, 0.9991370418689945, 1.0},
                          {0.0, 0.0, 0.0, 0.0, 0.0, 0.5042041932513908}};



  std::size_t bytes = a.size() * sizeof(Scalar);

  Scalar* d_a;
  Scalar* d_x;
  Scalar* d_out;
  hipMalloc((void**)(&d_a), bytes);
  hipMalloc((void**)(&d_x), bytes);
  hipMalloc((void**)(&d_out), bytes);

  hipMemcpy(d_a, a.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_x, x.data(), bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_a(d_a, 6, 6);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_x(d_x, 6, 6);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_out(d_out, 6, 6);

  gpu_out.device(gpu_device) = gpu_a.igamma(gpu_x);

  assert(hipMemcpyAsync(out.data(), d_out, bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 6; ++i) {
    for (int j = 0; j < 6; ++j) {
      if ((std::isnan)(igamma_s[i][j])) {
        VERIFY((std::isnan)(out(i, j)));
      } else {
        VERIFY_IS_APPROX(out(i, j), igamma_s[i][j]);
      }
    }
  }
}

template <typename Scalar>
void test_cuda_igammac()
{
  Tensor<Scalar, 2> a(6, 6);
  Tensor<Scalar, 2> x(6, 6);
  Tensor<Scalar, 2> out(6, 6);
  out.setZero();

  Scalar a_s[] = {Scalar(0), Scalar(1), Scalar(1.5), Scalar(4), Scalar(0.0001), Scalar(1000.5)};
  Scalar x_s[] = {Scalar(0), Scalar(1), Scalar(1.5), Scalar(4), Scalar(0.0001), Scalar(1000.5)};

  for (int i = 0; i < 6; ++i) {
    for (int j = 0; j < 6; ++j) {
      a(i, j) = a_s[i];
      x(i, j) = x_s[j];
    }
  }

  Scalar nan = std::numeric_limits<Scalar>::quiet_NaN();
  Scalar igammac_s[][6] = {{nan, nan, nan, nan, nan, nan},
                           {1.0, 0.36787944117144233, 0.22313016014842982,
                            0.018315638888734182, 0.9999000049998333, 0.0},
                           {1.0, 0.5724067044708798, 0.3916251762710878,
                            0.04601170568923136, 0.9999992477923555, 0.0},
                           {1.0, 0.9810118431238462, 0.9343575456215499,
                            0.4334701203667089, 1.0, 0.0},
                           {1.0, 2.1940638138146658e-05, 1.0003291916285e-05,
                            3.7801620118431334e-07, 0.0008629581310054535,
                            0.0},
                           {1.0, 1.0, 1.0, 1.0, 1.0, 0.49579580674813944}};

  std::size_t bytes = a.size() * sizeof(Scalar);

  Scalar* d_a;
  Scalar* d_x;
  Scalar* d_out;
  hipMalloc((void**)(&d_a), bytes);
  hipMalloc((void**)(&d_x), bytes);
  hipMalloc((void**)(&d_out), bytes);

  hipMemcpy(d_a, a.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_x, x.data(), bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_a(d_a, 6, 6);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_x(d_x, 6, 6);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_out(d_out, 6, 6);

  gpu_out.device(gpu_device) = gpu_a.igammac(gpu_x);

  assert(hipMemcpyAsync(out.data(), d_out, bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 6; ++i) {
    for (int j = 0; j < 6; ++j) {
      if ((std::isnan)(igammac_s[i][j])) {
        VERIFY((std::isnan)(out(i, j)));
      } else {
        VERIFY_IS_APPROX(out(i, j), igammac_s[i][j]);
      }
    }
  }
}

template <typename Scalar>
void test_cuda_erf(const Scalar stddev)
{
  Tensor<Scalar, 2> in(72,97);
  in.setRandom();
  in *= in.constant(stddev);
  Tensor<Scalar, 2> out(72,97);
  out.setZero();

  std::size_t bytes = in.size() * sizeof(Scalar);

  Scalar* d_in;
  Scalar* d_out;
  hipMalloc((void**)(&d_in), bytes);
  hipMalloc((void**)(&d_out), bytes);

  hipMemcpy(d_in, in.data(), bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_in(d_in, 72, 97);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_out(d_out, 72, 97);

  gpu_out.device(gpu_device) = gpu_in.erf();

  assert(hipMemcpyAsync(out.data(), d_out, bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 72; ++i) {
    for (int j = 0; j < 97; ++j) {
      VERIFY_IS_APPROX(out(i,j), (std::erf)(in(i,j)));
    }
  }

  hipFree(d_in);
  hipFree(d_out);
}

template <typename Scalar>
void test_cuda_erfc(const Scalar stddev)
{
  Tensor<Scalar, 2> in(72,97);
  in.setRandom();
  in *= in.constant(stddev);
  Tensor<Scalar, 2> out(72,97);
  out.setZero();

  std::size_t bytes = in.size() * sizeof(Scalar);

  Scalar* d_in;
  Scalar* d_out;
  hipMalloc((void**)(&d_in), bytes);
  hipMalloc((void**)(&d_out), bytes);

  hipMemcpy(d_in, in.data(), bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_in(d_in, 72, 97);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_out(d_out, 72, 97);

  gpu_out.device(gpu_device) = gpu_in.erfc();

  assert(hipMemcpyAsync(out.data(), d_out, bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 72; ++i) {
    for (int j = 0; j < 97; ++j) {
      VERIFY_IS_APPROX(out(i,j), (std::erfc)(in(i,j)));
    }
  }

  hipFree(d_in);
  hipFree(d_out);
}

void test_cxx11_tensor_cuda()
{
  CALL_SUBTEST_1(test_cuda_elementwise_small());
  CALL_SUBTEST_1(test_cuda_elementwise());
  CALL_SUBTEST_1(test_cuda_reduction());
  CALL_SUBTEST_2(test_cuda_contraction<ColMajor>());
  CALL_SUBTEST_2(test_cuda_contraction<RowMajor>());
  CALL_SUBTEST_3(test_cuda_convolution_1d<ColMajor>());
  CALL_SUBTEST_3(test_cuda_convolution_1d<RowMajor>());
  CALL_SUBTEST_3(test_cuda_convolution_inner_dim_col_major_1d());
  CALL_SUBTEST_3(test_cuda_convolution_inner_dim_row_major_1d());
  CALL_SUBTEST_3(test_cuda_convolution_2d<ColMajor>());
  CALL_SUBTEST_3(test_cuda_convolution_2d<RowMajor>());
  CALL_SUBTEST_3(test_cuda_convolution_3d<ColMajor>());
  CALL_SUBTEST_3(test_cuda_convolution_3d<RowMajor>());

#if __cplusplus > 199711L
  // std::erf, std::erfc, and so on where only added in c++11. We use them
  // as a golden reference to validate the results produced by Eigen. Therefore
  // we can only run these tests if we use a c++11 compiler.
  CALL_SUBTEST_4(test_cuda_lgamma<float>(1.0f));
  CALL_SUBTEST_4(test_cuda_lgamma<float>(100.0f));
  CALL_SUBTEST_4(test_cuda_lgamma<float>(0.01f));
  CALL_SUBTEST_4(test_cuda_lgamma<float>(0.001f));

  CALL_SUBTEST_4(test_cuda_digamma<float>());


  CALL_SUBTEST_4(test_cuda_erf<float>(1.0f));
  CALL_SUBTEST_4(test_cuda_erf<float>(100.0f));
  CALL_SUBTEST_4(test_cuda_erf<float>(0.01f));
  CALL_SUBTEST_4(test_cuda_erf<float>(0.001f));

  CALL_SUBTEST_4(test_cuda_erfc<float>(1.0f));
  // CALL_SUBTEST(test_cuda_erfc<float>(100.0f));
  CALL_SUBTEST_4(test_cuda_erfc<float>(5.0f)); // CUDA erfc lacks precision for large inputs
  CALL_SUBTEST_4(test_cuda_erfc<float>(0.01f));
  CALL_SUBTEST_4(test_cuda_erfc<float>(0.001f));

  CALL_SUBTEST_4(test_cuda_lgamma<double>(1.0));
  CALL_SUBTEST_4(test_cuda_lgamma<double>(100.0));
  CALL_SUBTEST_4(test_cuda_lgamma<double>(0.01));
  CALL_SUBTEST_4(test_cuda_lgamma<double>(0.001));

  CALL_SUBTEST_4(test_cuda_digamma<double>());

  CALL_SUBTEST_4(test_cuda_erf<double>(1.0));
  CALL_SUBTEST_4(test_cuda_erf<double>(100.0));
  CALL_SUBTEST_4(test_cuda_erf<double>(0.01));
  CALL_SUBTEST_4(test_cuda_erf<double>(0.001));

  CALL_SUBTEST_4(test_cuda_erfc<double>(1.0));
  // CALL_SUBTEST(test_cuda_erfc<double>(100.0));
  CALL_SUBTEST_4(test_cuda_erfc<double>(5.0)); // CUDA erfc lacks precision for large inputs
  CALL_SUBTEST_4(test_cuda_erfc<double>(0.01));
  CALL_SUBTEST_4(test_cuda_erfc<double>(0.001));

  CALL_SUBTEST_5(test_cuda_igamma<float>());
  CALL_SUBTEST_5(test_cuda_igammac<float>());

  CALL_SUBTEST_5(test_cuda_igamma<double>());
  CALL_SUBTEST_5(test_cuda_igammac<double>());
#endif
}
