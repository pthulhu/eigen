#include "hip/hip_runtime.h"
// This file is part of Eigen, a lightweight C++ template library
// for linear algebra.
//
// Copyright (C) 2016 Benoit Steiner <benoit.steiner.goog@gmail.com>
//
// This Source Code Form is subject to the terms of the Mozilla
// Public License v. 2.0. If a copy of the MPL was not distributed
// with this file, You can obtain one at http://mozilla.org/MPL/2.0/.

#define EIGEN_TEST_NO_LONGDOUBLE
#define EIGEN_TEST_FUNC cxx11_tensor_complex
#define EIGEN_USE_GPU

#if defined __CUDACC_VER__ && __CUDACC_VER__ >= 70500
#include <hip/hip_fp16.h>
#endif
#include "main.h"
#include <unsupported/Eigen/CXX11/Tensor>

using Eigen::Tensor;

void test_cuda_nullary() {
  Tensor<std::complex<float>, 1, 0, int> in1(2);
  Tensor<std::complex<float>, 1, 0, int> in2(2);
  in1.setRandom();
  in2.setRandom();

  std::size_t float_bytes = in1.size() * sizeof(float);
  std::size_t complex_bytes = in1.size() * sizeof(std::complex<float>);

  std::complex<float>* d_in1;
  std::complex<float>* d_in2;
  float* d_out2;
  hipMalloc((void**)(&d_in1), complex_bytes);
  hipMalloc((void**)(&d_in2), complex_bytes);
  hipMalloc((void**)(&d_out2), float_bytes);
  hipMemcpy(d_in1, in1.data(), complex_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_in2, in2.data(), complex_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<std::complex<float>, 1, 0, int>, Eigen::Aligned> gpu_in1(
      d_in1, 2);
  Eigen::TensorMap<Eigen::Tensor<std::complex<float>, 1, 0, int>, Eigen::Aligned> gpu_in2(
      d_in2, 2);
  Eigen::TensorMap<Eigen::Tensor<float, 1, 0, int>, Eigen::Aligned> gpu_out2(
      d_out2, 2);

  gpu_in1.device(gpu_device) = gpu_in1.constant(std::complex<float>(3.14f, 2.7f));
  gpu_out2.device(gpu_device) = gpu_in2.abs();

  Tensor<std::complex<float>, 1, 0, int> new1(2);
  Tensor<float, 1, 0, int> new2(2);

  assert(hipMemcpyAsync(new1.data(), d_in1, complex_bytes, hipMemcpyDeviceToHost,
                         gpu_device.stream()) == hipSuccess);
  assert(hipMemcpyAsync(new2.data(), d_out2, float_bytes, hipMemcpyDeviceToHost,
                         gpu_device.stream()) == hipSuccess);

  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 2; ++i) {
    VERIFY_IS_APPROX(new1(i), std::complex<float>(3.14f, 2.7f));
    VERIFY_IS_APPROX(new2(i), std::abs(in2(i)));
  }

  hipFree(d_in1);
  hipFree(d_in2);
  hipFree(d_out2);
}



void test_cxx11_tensor_complex()
{
  CALL_SUBTEST(test_cuda_nullary());
}
