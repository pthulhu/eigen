#include "hip/hip_runtime.h"
// This file is part of Eigen, a lightweight C++ template library
// for linear algebra.
//
// Copyright (C) 2016 Benoit Steiner <benoit.steiner.goog@gmail.com>
//
// This Source Code Form is subject to the terms of the Mozilla
// Public License v. 2.0. If a copy of the MPL was not distributed
// with this file, You can obtain one at http://mozilla.org/MPL/2.0/.

#define EIGEN_TEST_NO_LONGDOUBLE
#define EIGEN_TEST_NO_COMPLEX
#define EIGEN_TEST_FUNC cxx11_tensor_of_float16_cuda
#define EIGEN_DEFAULT_DENSE_INDEX_TYPE int
#define EIGEN_USE_GPU


#include "main.h"
#include <unsupported/Eigen/CXX11/Tensor>

using Eigen::Tensor;

#ifdef EIGEN_HAS_CUDA_FP16

void test_cuda_conversion() {
  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);
  int num_elem = 101;
  
  float* d_float = (float*)gpu_device.allocate(num_elem * sizeof(float));
  Eigen::half* d_half = (Eigen::half*)gpu_device.allocate(num_elem * sizeof(Eigen::half));
  float* d_conv = (float*)gpu_device.allocate(num_elem * sizeof(float));

  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_float(
      d_float, num_elem);
  Eigen::TensorMap<Eigen::Tensor<Eigen::half, 1>, Eigen::Aligned> gpu_half(
      d_half, num_elem);
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_conv(
      d_conv, num_elem);

  gpu_float.device(gpu_device) = gpu_float.random();
  gpu_half.device(gpu_device) = gpu_float.cast<Eigen::half>();
  gpu_conv.device(gpu_device) = gpu_half.cast<float>();

  Tensor<float, 1> initial(num_elem);
  Tensor<float, 1> final(num_elem);
  gpu_device.memcpyDeviceToHost(initial.data(), d_float, num_elem*sizeof(float));
  gpu_device.memcpyDeviceToHost(final.data(), d_conv, num_elem*sizeof(float));

  for (int i = 0; i < num_elem; ++i) {
    VERIFY_IS_APPROX(initial(i), final(i));
  }

  gpu_device.deallocate(d_float);
  gpu_device.deallocate(d_half);
  gpu_device.deallocate(d_conv);
}


void test_cuda_unary() {
  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);
  int num_elem = 101;

  float* d_float = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_res_half = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_res_float = (float*)gpu_device.allocate(num_elem * sizeof(float));

  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_float(
      d_float, num_elem);
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_res_half(
      d_res_half, num_elem);
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_res_float(
      d_res_float, num_elem);

  gpu_float.device(gpu_device) = gpu_float.random() - gpu_float.constant(0.5f);
  gpu_res_float.device(gpu_device) = gpu_float.abs();
  gpu_res_half.device(gpu_device) = gpu_float.cast<Eigen::half>().abs().cast<float>();

  Tensor<float, 1> half_prec(num_elem);
  Tensor<float, 1> full_prec(num_elem);
  gpu_device.memcpyDeviceToHost(half_prec.data(), d_res_half, num_elem*sizeof(float));
  gpu_device.memcpyDeviceToHost(full_prec.data(), d_res_float, num_elem*sizeof(float));
  gpu_device.synchronize();

  for (int i = 0; i < num_elem; ++i) {
    std::cout << "Checking unary " << i << std::endl;
    VERIFY_IS_APPROX(full_prec(i), half_prec(i));
  }

  gpu_device.deallocate(d_float);
  gpu_device.deallocate(d_res_half);
  gpu_device.deallocate(d_res_float);
}


void test_cuda_elementwise() {
  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);
  int num_elem = 101;

  float* d_float1 = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_float2 = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_res_half = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_res_float = (float*)gpu_device.allocate(num_elem * sizeof(float));

  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_float1(
      d_float1, num_elem);
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_float2(
      d_float2, num_elem);
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_res_half(
      d_res_half, num_elem);
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_res_float(
      d_res_float, num_elem);

  gpu_float1.device(gpu_device) = gpu_float1.random();
  gpu_float2.device(gpu_device) = gpu_float2.random();
  gpu_res_float.device(gpu_device) = (gpu_float1 + gpu_float2) * gpu_float1;
  gpu_res_half.device(gpu_device) = ((gpu_float1.cast<Eigen::half>() + gpu_float2.cast<Eigen::half>()) * gpu_float1.cast<Eigen::half>()).cast<float>();

  Tensor<float, 1> half_prec(num_elem);
  Tensor<float, 1> full_prec(num_elem);
  gpu_device.memcpyDeviceToHost(half_prec.data(), d_res_half, num_elem*sizeof(float));
  gpu_device.memcpyDeviceToHost(full_prec.data(), d_res_float, num_elem*sizeof(float));
  gpu_device.synchronize();

  for (int i = 0; i < num_elem; ++i) {
    std::cout << "Checking elemwise " << i << std::endl;
    VERIFY_IS_APPROX(full_prec(i), half_prec(i));
  }

  gpu_device.deallocate(d_float1);
  gpu_device.deallocate(d_float2);
  gpu_device.deallocate(d_res_half);
  gpu_device.deallocate(d_res_float);
}


void test_cuda_contractions() {
  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);
  int rows = 23;
  int cols = 23;
  int num_elem = rows*cols;

  float* d_float1 = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_float2 = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_res_half = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_res_float = (float*)gpu_device.allocate(num_elem * sizeof(float));

  Eigen::TensorMap<Eigen::Tensor<float, 2>, Eigen::Aligned> gpu_float1(
      d_float1, rows, cols);
  Eigen::TensorMap<Eigen::Tensor<float, 2>, Eigen::Aligned> gpu_float2(
      d_float2, rows, cols);
  Eigen::TensorMap<Eigen::Tensor<float, 2>, Eigen::Aligned> gpu_res_half(
      d_res_half, rows, cols);
  Eigen::TensorMap<Eigen::Tensor<float, 2>, Eigen::Aligned> gpu_res_float(
      d_res_float, rows, cols);

  gpu_float1.device(gpu_device) = gpu_float1.random() - gpu_float1.constant(0.5f);
  gpu_float2.device(gpu_device) = gpu_float2.random() - gpu_float1.constant(0.5f);

  typedef Tensor<float, 2>::DimensionPair DimPair;
  Eigen::array<DimPair, 1> dims(DimPair(1, 0));
  gpu_res_float.device(gpu_device) = gpu_float1.contract(gpu_float2, dims);
  gpu_res_half.device(gpu_device) = gpu_float1.cast<Eigen::half>().contract(gpu_float2.cast<Eigen::half>(), dims).cast<float>();

  Tensor<float, 2> half_prec(rows, cols);
  Tensor<float, 2> full_prec(rows, cols);
  gpu_device.memcpyDeviceToHost(half_prec.data(), d_res_half, num_elem*sizeof(float));
  gpu_device.memcpyDeviceToHost(full_prec.data(), d_res_float, num_elem*sizeof(float));
  gpu_device.synchronize();

  for (int i = 0; i < rows; ++i) {
    for (int j = 0; j < cols; ++j) {
      std::cout << "Checking contract " << i << " " << j << std::endl;
      VERIFY_IS_APPROX(full_prec(i, j), half_prec(i, j));
    }
  }

  gpu_device.deallocate(d_float1);
  gpu_device.deallocate(d_float2);
  gpu_device.deallocate(d_res_half);
  gpu_device.deallocate(d_res_float);
}


void test_cuda_reductions() {
  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);
  int size = 13;
  int num_elem = size*size;

  float* d_float1 = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_float2 = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_res_half = (float*)gpu_device.allocate(size * sizeof(float));
  float* d_res_float = (float*)gpu_device.allocate(size * sizeof(float));

  Eigen::TensorMap<Eigen::Tensor<float, 2>, Eigen::Aligned> gpu_float1(
      d_float1, size, size);
  Eigen::TensorMap<Eigen::Tensor<float, 2>, Eigen::Aligned> gpu_float2(
      d_float2, size, size);
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_res_half(
      d_res_half, size);
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_res_float(
      d_res_float, size);

  gpu_float1.device(gpu_device) = gpu_float1.random();
  gpu_float2.device(gpu_device) = gpu_float2.random();

  Eigen::array<int, 1> redux_dim = {{0}};
  gpu_res_float.device(gpu_device) = gpu_float1.sum(redux_dim);
  gpu_res_half.device(gpu_device) = gpu_float1.cast<Eigen::half>().sum(redux_dim).cast<float>();

  Tensor<float, 1> half_prec(size);
  Tensor<float, 1> full_prec(size);
  gpu_device.memcpyDeviceToHost(half_prec.data(), d_res_half, size*sizeof(float));
  gpu_device.memcpyDeviceToHost(full_prec.data(), d_res_float, size*sizeof(float));
  gpu_device.synchronize();

  for (int i = 0; i < size; ++i) {
    std::cout << "Checking redux " << i << std::endl;
    VERIFY_IS_APPROX(full_prec(i), half_prec(i));
  }

  gpu_device.deallocate(d_float1);
  gpu_device.deallocate(d_float2);
  gpu_device.deallocate(d_res_half);
  gpu_device.deallocate(d_res_float);
}


#endif


void test_cxx11_tensor_of_float16_cuda()
{
#ifdef EIGEN_HAS_CUDA_FP16
  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice device(&stream);
  if (device.majorDeviceVersion() > 5 ||
      (device.majorDeviceVersion() == 5 && device.minorDeviceVersion() >= 3)) {
    std::cout << "Running test on device with capability " << device.majorDeviceVersion() << "." << device.minorDeviceVersion() << std::endl;

    CALL_SUBTEST_1(test_cuda_conversion());
    CALL_SUBTEST_1(test_cuda_unary());
    CALL_SUBTEST_1(test_cuda_elementwise());
    CALL_SUBTEST_2(test_cuda_contractions());
    CALL_SUBTEST_3(test_cuda_reductions());
  }
  else {
   std::cout << "Half floats require compute capability of at least 5.3. This device only supports " << device.majorDeviceVersion() << "." << device.minorDeviceVersion() << ". Skipping the test" << std::endl;
  }
#else
  std::cout << "Half floats are not supported by this version of cuda: skipping the test" << std::endl;
#endif
}
